#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

#define MAX_THREADS 256

__global__ void vectorAdd(float *cudaA, float *cudaB, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        cudaA[i] += cudaB[i];
    }
    return;
}

void gpu_addPositionMatrix(float *cuda_embed_matrix, float *cuda_position_matrix, int embed_elements)
{
    int threads = 256;
    int blocks = (embed_elements + threads - 1) / threads;
    vectorAdd<<<blocks, threads>>>(cuda_embed_matrix, cuda_position_matrix, embed_elements);
    hipDeviceSynchronize();
    return;
}

// m = rows of A
// k = cols of A = rows of B
// n = cols of B
// C = result
void gpu_cublas_matmul(float *A, float *B, float *C, int m, int k, int n)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        A, m,
        B, k,
        &beta,
        C, m
    );

    hipblasDestroy(handle);
    return;
}

void gpu_init_zero(float **pointer, int elements)
{
    float *zero = (float*)calloc(elements, sizeof(float));
    if(zero == NULL) exit(EXIT_FAILURE);
    gpu_init(pointer, zero, elements);
    free(zero);
    return;
}

void gpu_init(float **pointer, float *matrix, int elements)
{
    size_t bytes = elements * sizeof(float);
    hipMalloc(pointer, bytes);
    hipMemcpy(*pointer, matrix, bytes, hipMemcpyHostToDevice);
    return;
}

void kernel_cudafree(float *pointer) 
{
    hipFree(pointer);
    return;
}

void kernel_memcpydevicetohost(float *dst, float *src, int size)
{
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    return;
}