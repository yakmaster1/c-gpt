#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void vectorAdd(float *cudaA, float *cudaB, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        cudaA[i] += cudaB[i];
    }
    return;
}

extern "C" void gpu_addvectors(float *matrix_a, float *matrix_b, int size)
{
    float *cudaA = 0;
    float *cudaB = 0;

    size_t bytes = size * sizeof(float);

    hipMalloc(&cudaA, bytes);
    hipMalloc(&cudaB, bytes);

    hipMemcpy(cudaA, matrix_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, matrix_b, bytes, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    vectorAdd<<<blocks, threads>>>(cudaA, cudaB, size);
    hipDeviceSynchronize();
    
    hipMemcpy(matrix_a, cudaA, bytes, hipMemcpyDeviceToHost);

    hipFree(cudaA);
    hipFree(cudaB);
}